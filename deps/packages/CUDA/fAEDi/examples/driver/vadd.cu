
#include <hip/hip_runtime.h>
extern "C" {

__global__ void kernel_vadd(const float *a, const float *b, float *c)
{
    int i = blockIdx.x *blockDim.x + threadIdx.x;
    c[i] = a[i] + b[i];
}

}
